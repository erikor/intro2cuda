
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <string>
#include <stdio.h>
#include <time.h>

#define N 1000

int main() {
	int i, j, n, value;
	int* a;
	int* b;
	srand(time(NULL));   // should only be called once
	int size = N * N * sizeof(int);

    a = (int*)malloc(size);
    b = (int*)malloc(size);

	// print out a bit of data to show starting values
	for(i = 0; i<10; i++) {
		printf("%d\n", b[i]);
	}	


    for(i=0; i<N; i++) {
    	for(j=0; j<N; j++) {
    		a[i+j*N] = rand();
    		//a[i+j*N] = 2;
    	}
    }
    for(i=0; i<N; i++) {
    	for(j=0; j<N; j++) {
    		for(n=0; n<1000; n++) {
		       value = a[i + j * N] + i;
    		}
    		b[i+j*N] = value;
    	}
    }

	// print out a bit of data to show we did something
	for(i = 0; i<10; i++) {
		printf("%d\n", b[i]);
	}	

	free(a);
	free(b);
	
	return 0;

}

