// demo 2: push some data to device

// compile with:
// nvcc demo2.cu -lcuda -o demo2

#include <stdio.h>
#include <hip/hip_runtime.h>

int main()
{
	int a[4] = {1,2,3,4};
    size_t size = sizeof(int) * 4;
	hipDeviceptr_t ad;
	hipMalloc( &ad, size);
    hipMemcpyHtoD(ad, a, size);
	hipFree(ad);
	printf("Success!\n");
	return 0;
}