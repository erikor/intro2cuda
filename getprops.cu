// retrieve device capabilities
// nvcc demo4.cu -o demo4
// time ./demo4c 

#include <iostream>
#include <string>
#include <typeinfo>
#include <hip/hip_runtime.h>

using namespace std;

/*
 *  Infer number of cores per multiprocessor from compute capability
 *  Based on Robert Crovella's SO response: https://goo.gl/XguKMK
 *  Also see https://people.maths.ox.ac.uk/gilesm/cuda/prac1/helper_cuda.h
 */
int getSPcores(hipDeviceProp_t devProp)
{  
  int cores = 0;
  switch (devProp.major){
  case 2: // Fermi
    if (devProp.minor == 1) cores = 48;
    else cores = 32;
    break;
  case 3: // Kepler
    cores = 192;
    break;
  case 5: // Maxwell
    cores = 128;
    break;
  case 6: // Pascal
    if (devProp.minor == 1) cores = 128;
    else if (devProp.minor == 0) cores = 64;
    else cores = -999;
    break;
  default:
    cores = -999;
  break;
  }
  return cores;
}

int main() {
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  cout << "DEVICE SPECIFICATIONS:" << endl << "----------------------" << endl;
  cout << "Device name: " << props.name << endl;
  cout << "Total Global Memory: " << props.totalGlobalMem << endl;
  cout << "Share mem per block: " << props.sharedMemPerBlock << endl;
  cout << "Registers per block: " << props.regsPerBlock << endl;
  cout << "Warp size: " << props.warpSize << endl;
  cout << "Max threads per block: " << props.maxThreadsPerBlock << endl;
  cout << "Max thread dimensions (x, y, z): " << props.maxThreadsDim[0] << ", "
       << props.maxThreadsDim[1] << ", "
       << props.maxThreadsDim[2] << endl;
  cout << "Max grid dimensions (x, y, z): " << props.maxGridSize[0] << ", "
       << props.maxGridSize[1] << ", "
       << props.maxGridSize[2] << endl;
  cout << "Compute capability: " << props.major << "." << props.minor << endl;
  cout << "Multiprocessor count: " << props.multiProcessorCount << endl;
  cout << "Cores per multiprocessor (inferred): " << getSPcores(props) << endl;
  return 0;
}