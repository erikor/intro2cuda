// demo 1: allocate and free device memory

// a note about headers:
// https://devtalk.nvidia.com/default/topic/1002219/does-nvcc-include-header-files-automatically-/

// build:
// nvcc demo1.cu -o demo1


#include <hip/hip_runtime.h>
#include <stdio.h>
 
int main()
{
	char *mymem;
	hipMalloc( (void**)&mymem, 1);
	hipFree(mymem);
	printf("Success!\n");
	return 0;
}