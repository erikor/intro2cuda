#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1000

__global__ void matrixMult (int *a, int *b, int *c, int width) {
	int k, sum = 0;
	long col = threadIdx.x + blockDim.x * blockIdx.x;
	long row = threadIdx.y + blockDim.y * blockIdx.y;
	if(col < width && row < width) {
		for (k = 0; k < width; k++)
			sum += a[row * width + k] * b[k * width + col];
		c[row * width + col] = sum;
	}
}

int main() {
	int i, j;
	int* a;
	int* b;
	int* c;
	hipDeviceptr_t ad, bd, cd;
	long size = N * N * sizeof(int);
	dim3 dimGrid(100, 100);
	dim3 dimBlock(100, 100);

    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);

    for(i=0; i<N; i++) {
    	for(j=0; j<N; j++) {
    		a[i+j*N] = 2;
    		b[i+j*N] = 2;
    		c[i+j*N] = 2;
    	}
    }

    hipDeviceSynchronize(); 

	hipMalloc(&ad, size);
	hipMalloc(&bd, size);
	hipMalloc(&cd, size);
	hipMemcpyHtoD(ad, a, size);
	hipMemcpyHtoD(bd, b, size);
	hipMemcpyHtoD(cd, c, size);

    hipDeviceSynchronize(); 

	//matrixMult <<<dimGrid, dimBlock>>>((int*)ad, (int*)bd, (int*)cd, N);

	//hipMemcpyDtoH(c, cd, size);
	hipFree(ad); 	hipFree(bd); 	hipFree(cd);

}

