// single threaded matrix multiply
// nvcc demo4.cu -o demo4
// time ./demo4


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 800

void matrixMult (int* a, int* b, int* c, int width)
{
	for (int i = 0; i < width; i++) {
		for (int j = 0; j < width; j++) {
			int sum = 0;
			for (int k = 0; k < width; k++) {
				int m = a[i + k * width];
				int n = b[k * width + j];
				sum += m * n;
			}
			c[i + width * j] = sum;
		}
	}
}

int main() {
	int i, j;
	int* a;
	int* b;
	int* c;
	long size = N * N * sizeof(int);
    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);

/*    for(i=0; i<N; i++) {
    	for(j=0; j<N; j++) {
    		a[i+j*N] = 2;
    		b[i+j*N] = 2;
    		c[i+j*N] = 2;
    	}
    }
*/
	matrixMult(a, b, c, N);
}
