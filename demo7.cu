#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <string>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include "safecalls.h"

using namespace std;

#define N 1000

__global__ void matrixShared (int *a, int *b, int width) {
    __shared__ int s[100 * sizeof(int)];
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int value;
	int col = threadIdx.x + blockDim.x * blockIdx.y;
	int row = threadIdx.y + blockDim.y * blockIdx.y;
    int i;

    s[tx + ty * blockDim.x] = a[col + row * width];	

    __syncthreads();
    
    for(i=0; i<1000; i++) {
       value = s[tx + ty * blockDim.x] + i;
    }
    a[col + row * width] = value;	
}

int main() {
	int i, j;
	int* a;
	int* b;
	srand(time(NULL)); 

	hipDeviceptr_t ad, bd;
	int size = N * N * sizeof(int);

	dim3 dimGrid(100, 100);
	dim3 dimBlock(10, 10);

    a = (int*)malloc(size);
    b = (int*)malloc(size);

    for(i=0; i<N; i++) {
    	for(j=0; j<N; j++) {
    		a[i+j*N] = rand();
    	}
    }

	// print out a bit of data to show starting values
	for(i = 0; i<10; i++) {
		printf("%d\n", b[i]);
	}	


	CUDA_SAFE_CALL(hipMalloc(&ad, size));
	CUDA_SAFE_CALL(hipMalloc(&bd, size));
	CUDA_SAFE_CALL(hipMemcpyHtoD(ad, a, size));

	matrixShared<<<dimGrid, dimBlock>>>((int*)ad, (int*)bd, N);

	CUDA_SAFE_CALL(hipMemcpyDtoH(b, bd, size));
	hipFree(ad); 	hipFree(bd);

	// print out a bit of data to show we did something
	for(i = 0; i<10; i++) {
		printf("%d\n", b[i]);
	}	
	return 0;

}

