#include "hip/hip_runtime.h"
// demo 3: push some data to device
// a first kernel
// compile with:
// nvcc demo3.cu -lcuda -o demo3
#include <iostream>
#include <string>
#include <typeinfo>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

using namespace std;
#include "safecalls.h"

__global__ void mykernel(int ncol, int nrow, int* input, int* output)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int index = idx + idy * ncol;
  if(index < ncol * nrow) {
    output[index] = input[index]; 	
  }
}

int main()
{
    dim3 dimBlock(4, 4); 
    dim3 dimGrid(1, 1); 
    size_t size = sizeof(int) * 16;
	int* in = (int*)malloc(size);
	int* out = (int*)malloc(size);
	hipDeviceptr_t ind;
	hipDeviceptr_t outd;
	int i;

    // initalize input and output 'matrices'
	for(i = 0; i < 16; i++) {
      in[i] = i;
	}

    cout << "Contents of out before kernel call:\n";
	for(i = 0; i < 16; i++) {
      cout << out[i] << " ";
	}
    cout << "\n\n";

	hipDeviceSynchronize(); 

	hipMalloc(&ind, size);
	hipMalloc(&outd, size);
	hipMemcpyHtoD(outd, out, size);
	hipMemcpyHtoD(ind, in, size);
	mykernel <<<dimGrid, dimBlock>>>(4, 4, (int*)ind, (int*)outd);
	hipMemcpyDtoH(out, outd, size);


    cout << "Contents of out after  kernel call:\n";
	for(i = 0; i < 16; i++) {
      cout << out[i] << " ";
	}
    cout << "\n";

	hipFree(ind);
	hipFree(outd);
	return 0;
}

